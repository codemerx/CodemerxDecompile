#include <stdlib.h>
#include <stdio.h>
#include <unistd.h>

#include <hip/hip_runtime.h>

#if defined(assert)
#undef assert
#endif

#define assert(c) \
    do { \
        if(!(c)) { \
            fprintf(stderr, "Assertion \"%s\" failed. (%s:%d)\n", \
                #c, __FILE__, __LINE__); \
            exit(1); \
        } \
    } while(0)

#define assertSucceeded(c) \
    do { \
        unsigned __tmp = c; \
        if(__tmp != hipSuccess) { \
            fprintf(stderr, "Operation \"%s\" failed with error code %x. (%s:%d)\n", \
                #c, (__tmp), __FILE__, __LINE__); \
            exit(__tmp); \
        } \
    } while(0)

#define ARRAY_LENGTH(x) (sizeof(x) / sizeof(x[0]))

constexpr int dataLength = 1 << 24;
constexpr int threadsPerBlock = 128;

typedef unsigned char byte;

struct TestType
{
    union {
        struct
        {
            unsigned lowHalf;
            unsigned highHalf;
        } halfAndHalf;

        unsigned long long whole;
    } takeYourPick;

    int arr[5];

    struct {
        char a;
        char b;
    } structArr[5];

    float theFloats[2];
    double theDouble;
};

__global__ void cudaComputeHash(TestType* input, unsigned *results)
{
    int idx = blockIdx.x * threadsPerBlock + threadIdx.x;
    TestType* myInput = input + idx;

    unsigned myResult = 0;

    myResult += myInput->takeYourPick.halfAndHalf.lowHalf - idx;
    myResult += myInput->takeYourPick.halfAndHalf.highHalf - idx;

    for(size_t i = 0; i < ARRAY_LENGTH(myInput->arr); i++)
    {
        myResult += myInput->arr[i] - idx;
    }

    for(size_t i = 0; i < sizeof(myInput->structArr); i++)
    {
        myResult += reinterpret_cast<byte *>(myInput->structArr)[i] - '0';
    }

    __syncthreads();

    results[idx] = myResult;
}

int main()
{
    int cudaDeviceCount;
    assertSucceeded(hipGetDeviceCount(&cudaDeviceCount));
    assert(cudaDeviceCount > 0);

    assertSucceeded(hipSetDevice(0));

    TestType* input;
    unsigned* results;

    assertSucceeded(hipMallocManaged(&input, sizeof(TestType) * dataLength));
    assert(!!input);

    for (size_t i = 0; i < dataLength; i++)
    {
        input[i].takeYourPick.halfAndHalf.lowHalf = i + 1;
        input[i].takeYourPick.halfAndHalf.highHalf = i + 3;

        for(size_t j = 0; j < ARRAY_LENGTH(input[i].arr); j++)
        {
            input[i].arr[j] = i + j + 2;
        }

        for(size_t j = 0; j < sizeof(input[i].structArr); j++)
        {
            reinterpret_cast<byte *>(input[i].structArr)[j] = '0' + static_cast<char>((i + j) % 10);
        }

        input[i].theFloats[0] = i + 1;
        input[i].theFloats[1] = input[i].theFloats[0] / 2;

        input[i].theDouble = input[i].theFloats[1] + 1;
    }

    assertSucceeded(hipMallocManaged(reinterpret_cast<void **>(&results), sizeof(unsigned) * dataLength));
    assert(!!results);

    constexpr int blocks = dataLength / threadsPerBlock;
    cudaComputeHash<<<blocks, threadsPerBlock>>>(input, results);

    assertSucceeded(hipDeviceSynchronize());

    const unsigned expectedResult =
        1 +
        3 +
        ARRAY_LENGTH(input[0].arr) * (ARRAY_LENGTH(input[0].arr) - 1) / 2 +
        ARRAY_LENGTH(input[0].arr) * 2 +
        sizeof(input[0].structArr) * (sizeof(input[0].structArr) - 1) / 2;

    for (unsigned i = 0; i < dataLength; i++)
    {
        if (results[i] != expectedResult){
            fprintf(stderr, "results[%u] (%u) != %u\n", i, results[i], expectedResult);
            exit(1);
        }
    }

    assertSucceeded(hipFree(input));
    assertSucceeded(hipFree(results));

    fprintf(stderr, "Success\n");

    exit(0);
}
